#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
extern "C" {
  #include "dim3_sweep_cuda.h"
}

// Local variable array macro
#define PSI_1D(ANG)   psi[ANG]
#define PC_1D(ANG)    pc[ANG]
#define DEN_1D(ANG)   den[ANG]

#ifdef ROWORDER
#define HV_2D(ANG, X) hv[ ANG*4                 \
                          + X ]
#else
#define HV_2D(ANG, X) hv[ X*NANG                \
                          + ANG ]
#endif

#ifdef ROWORDER
#define FXHV_2D(ANG, X) fxhv[ ANG*4             \
                              + X ]
#else
#define FXHV_2D(ANG, X) fxhv[ X*NANG            \
                              + ANG ]
#endif

// Simplify array indexing when certain values constant throughout module
#define PSII_3D(ANG, Y, Z)       PSII_4D(ANG, Y, Z, (g-1))
#define PSIJ_3D(ANG, CHUNK, Z)   PSIJ_4D(ANG, CHUNK, Z, (g-1))
#define PSIK_3D(ANG, CHUNK, Y)   PSIK_4D(ANG, CHUNK, Y, (g-1))
#define QTOT_4D(MOM1, X, Y, Z)   QTOT_5D(MOM1, X, Y, Z, (g-1))
#define EC_2D(ANG, MOM1)         EC_3D(ANG, MOM1, (oct-1))
#define VDELT_CONST              VDELT_1D(g-1)
#define PTR_IN_4D(ANG, X, Y, Z)  PTR_IN_6D(ANG, X, Y, Z, (i1-1), (i2-1))
#define PTR_OUT_4D(ANG, X, Y, Z) PTR_OUT_6D(ANG, X, Y, Z, (i1-1), (i2-1))
#define DINV_4D(ANG, X, Y, Z)    DINV_5D(ANG, X, Y, Z, (g-1))
#define FLUX_3D(X, Y, Z)         FLUX_4D(X, Y, Z, (g-1))
#define FLUXM_4D(MOM1, X, Y, Z)  FLUXM_5D(MOM1, X, Y, Z, (g-1))
#define JB_IN_3D(ANG, CHUNK, Z)  JB_IN_4D(ANG, CHUNK, Z, (g-1))
#define JB_OUT_3D(ANG, CHUNK, Z) JB_OUT_4D(ANG, CHUNK, Z, (g-1))
#define KB_IN_3D(ANG, CHUNK, Y)  KB_IN_4D(ANG, CHUNK, Y, (g-1))
#define KB_OUT_3D(ANG, CHUNK, Y) KB_OUT_4D(ANG, CHUNK, Y, (g-1))
#define FLKX_3D(X, Y, Z)         FLKX_4D(X, Y, Z, (g-1))
#define FLKY_3D(X, Y, Z)         FLKY_4D(X, Y, Z, (g-1))
#define FLKZ_3D(X, Y, Z)         FLKZ_4D(X, Y, Z, (g-1))
#define T_XS_3D(X, Y, Z)         T_XS_4D(X, Y, Z, (g-1))


// CUDA vars
#define N   10



__global__ void add( int *a, int *b, int *c )
{
    int tid = blockIdx.x;    // this thread handles the data at its thread id
    if (tid < N)
        c[tid] = a[tid] + b[tid];
}

int test( void )
{
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    // allocate the memory on the GPU
    hipMalloc( (void**)&dev_a, N * sizeof(int) );
    hipMalloc( (void**)&dev_b, N * sizeof(int) );
    hipMalloc( (void**)&dev_c, N * sizeof(int) );

    // fill the arrays 'a' and 'b' on the CPU
    for (int i=0; i<N; i++) {
        a[i] = -i;
        b[i] = i * i;
    }

    // copy the arrays 'a' and 'b' to the GPU
    hipMemcpy( dev_a, a, N * sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy( dev_b, b, N * sizeof(int), hipMemcpyHostToDevice );

    add<<<N,1>>>( dev_a, dev_b, dev_c );

    // copy the array 'c' back from the GPU to the CPU
    hipMemcpy( c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost );

    // display the results
    for (int i=0; i<N; i++) {
        printf( "%d + %d = %d\n", a[i], b[i], c[i] );
    }

    // free the memory allocated on the GPU
    hipFree( dev_a );
    hipFree( dev_b );
    hipFree( dev_c );

    return 0;
}

__global__
void diagonal_loop( input_data *input_vars, 
    bool firsty, bool lasty, bool firstz, bool lastz,
    geom_data *geom_vars, sn_data *sn_vars,
    data_data *data_vars, control_data *control_vars,
    solvar_data *solvar_vars, dim_sweep_data *dim_sweep_vars,

    int ich, int i_dir, int d1, int d2, int d3, int d4, int j_dir,
    int k_dir, int j_low, int k_low, int j_high, int k_high, int j_step, int k_step,
    int i1, int i2, int oct, int g, int *ierr,

    int nang, 
    double *psi, double *pc, double *den, double *hv, double *fxhv,

    int d)
{
    // Local variables
    int n = threadIdx.x;
    int i_step, ic, i, j, k, l, ibl, ibb, ibf;

    int ang, indx1 = 4;

    double sum_hv = 0, sum_hv_tmp = 0, sum_wpsi = 0, sum_ecwpsi = 0,
        sum_wmupsii = 0, sum_wetapsij = 0, sum_wxipsik = 0;

    // Set up the sweep order in the i-direction.
    i_step = -1;
    if ( i_dir == 2 ) i_step = 1;

    // Loop over cells along the diagonals. When only 1 diagonal, it's
    // normal sweep order. Otherwise, nested threading performs mini-KBA.
    ic = DIAG_1D(d-1).cell_id_vars[n-1].ic;

    if ( i_step < 0 )
    {
        i = ich*ICHUNK - ic + 1;
    }
    else
    {
        i = (ich-1)*ICHUNK + ic;
    }

    if ( i <= NX )
    {
        j = DIAG_1D(d-1).cell_id_vars[n-1].jc;

        if ( j_step < 0 )
        {
            j = NY - j + 1;
        }

        k = DIAG_1D(d-1).cell_id_vars[n-1].kc;

        if ( k_step < 0 )
        {
            k = NZ - k + 1;
        }

        // Left/right boundary conditions, always vacuum.
        ibl = 0;

        if ( (i == NX) && (i_step == -1) )
        {
            for ( ang = 0; ang < nang; ang++ )
            {
                PSII_3D(ang,(j-1),(k-1)) = 0;
            }
        }
        else if ( i == 1 && i_step == 1 )
        {
            switch ( ibl )
            {
            case 0:
            case 1:
                for ( ang = 0; ang < nang; ang++ )
                {
                    PSII_3D(ang,(j-1),(k-1)) = 0;
                }
            }
        }

        // Top/bottom boundary condtions. Vacuum at global boundaries,
        // but set to some incoming flux from neighboring proc.
        ibb = 0;
        
        if ( j == j_low )
        {
            if ( j_dir == 1 && lasty )
            {
                for ( ang = 0; ang < nang; ang++ )
                {
                    PSIJ_3D(ang,(ic-1),(k-1)) = 0;
                }
            }
            else if ( j_dir == 2 && firsty )
            {
                switch ( ibb )
                {
                case 0:
                case 1:
                    for ( ang = 0; ang < nang; ang++ )
                    {
                        PSIJ_3D(ang,(ic-1),(k-1)) = 0;
                    }
                }
            }
            else
            {
                for ( ang = 0; ang < nang; ang++ )
                {
                    PSIJ_3D(ang,(ic-1),(k-1))
                        = JB_IN_3D(ang,(ic-1),(k-1));
                }
            }
        }

        // Front/back boundary condtions. Vacuum at global boundaries, 
        // but set to some incoming flux from neighboring proc.
        ibf = 0;
        
        if ( k == k_low )
        {
            if ( (k_dir == 1 && lastz) || NDIMEN < 3 )
            {
                for ( ang = 0; ang < nang; ang++ )
                {
                    PSIK_3D(ang,(ic-1),(j-1)) = 0;
                }
            }
            else if ( k_dir == 2 && firstz )
            {
                switch ( ibf )
                {
                case 0:
                case 1:
                    for ( ang = 0; ang < nang; ang++ )
                    {
                        PSIK_3D(ang,(ic-1),(j-1)) = 0;
                    }
                }
            }
            else
            {
                for ( ang = 0; ang < nang; ang++ )
                {
                    PSIK_3D(ang,(ic-1),(j-1))
                        = KB_IN_3D(ang,(ic-1),(j-1));
                }
            }
        }

        // Compute the angular source
        for ( ang = 0; ang < nang; ang++ )
        {
            PSI_1D(ang) = QTOT_4D(0,(i-1),(j-1),(k-1));

            if ( SRC_OPT == 3 )
            {
                PSI_1D(ang) +=
                    QIM_6D(ang,(i-1),(j-1),(k-1),(oct-1),(g-1));
            }
        }

        for ( l = 2; l <= CMOM; l++ )
        {
            for ( ang = 0; ang < nang; ang++ )
            {
                PSI_1D(ang) +=
                    EC_2D(ang,(l-1))
                    *QTOT_4D((l-1),(i-1),(j-1),(k-1));
            }
        }

        // Compute the numerator for the update formula
        for ( ang = 0; ang < nang; ang++ )
        {
            PC_1D(ang) = PSI_1D(ang)
                + PSII_3D(ang,(j-1),(k-1)) *MU_1D(ang)*HI
                + PSIJ_3D(ang,(ic-1),(k-1))*HJ_1D(ang)
                + PSIK_3D(ang,(ic-1),(j-1))*HK_1D(ang);

            if ( VDELT_CONST != 0 )
            {
                PC_1D(ang) += VDELT_CONST
                    *PTR_IN_6D(ang,(i-1),(j-1),(k-1),(i1-1),(i2-1));
            }
        }

        // Compute the solution of the center. Use DD for edges. 
        // Use fixup if requested.
        if ( FIXUP == 0 )
        {
            for ( ang = 0; ang < nang; ang++ )
            {
                PSI_1D(ang)
                    = PC_1D(ang)*DINV_4D(ang,(i-1),(j-1),(k-1));

                PSII_3D(ang,(j-1),(k-1))
                    = 2*PSI_1D(ang) - PSII_3D(ang,(j-1),(k-1));

                PSIJ_3D(ang,(ic-1),(k-1))
                    = 2*PSI_1D(ang) - PSIJ_3D(ang,(ic-1),(k-1));

                if ( NDIMEN == 3 )
                {
                    PSIK_3D(ang,(ic-1),(j-1))
                        = 2*PSI_1D(ang) - PSIK_3D(ang,(ic-1),(j-1));
                }

                if ( VDELT_CONST != 0 )
                {
                    PTR_OUT_6D(ang,(i-1),(j-1),(k-1),(i1-1),(i2-1))
                        = 2*PSI_1D(ang)
                        - PTR_IN_6D(ang,(i-1),(j-1),(k-1),(i1-1),(i2-1));
                }
            }
        }
        else
        {
            // Multi-pass set to zero + rebalance fixup. Determine angles
            // that will need fixup first.
            sum_hv = 0;
            for (ang = 0; ang < nang; ang++)
            {
                for (indx1 = 0; indx1 < 4; indx1++)
                {
                    HV_2D(ang, indx1) = 1;
                    sum_hv += HV_2D(ang,indx1);
                }

                PC_1D(ang) = PC_1D(ang) * DINV_4D(ang,(i-1),(j-1),(k-1));
            }

            // fixup_loop
            while (true)
            {
                sum_hv_tmp = 0;

                for ( ang = 0; ang < nang; ang++ )
                {
                    FXHV_2D(ang,0) =  2*PC_1D(ang) - PSII_3D(ang,(j-1),(k-1));

                    FXHV_2D(ang,1) =  2*PC_1D(ang) - PSIJ_3D(ang,(ic-1),(k-1));

                    if ( NDIMEN == 3 )
                    {
                        FXHV_2D(ang,2) = 2*PC_1D(ang) - PSIK_3D(ang,(ic-1),(j-1));
                    }

                    if ( VDELT_CONST != 0 )
                    {
                        FXHV_2D(ang,3) = 2*PC_1D(ang) - PTR_IN_6D(ang,(i-1),(j-1),(k-1),(i1-1),(i2-1));
                    }

                    for ( indx1 = 0; indx1 < 4; indx1++ )
                    {
                        if ( FXHV_2D(ang,indx1) < 0 )
                        {
                            HV_2D(ang,indx1) = 0;
                        }
                        sum_hv_tmp += HV_2D(ang,indx1);
                    }
                }

                // Exit loop when all angles are fixed up
                if (sum_hv == sum_hv_tmp) break;

                sum_hv = sum_hv_tmp;

                // Recompute balance equation numerator and denominator 
                // and get new cell average flux
                for ( ang = 0; ang < nang; ang++ )
                {
                    PC_1D(ang) = PSII_3D(ang,(j-1),(k-1))
                        * MU_1D(ang) * HI * (1+HV_2D(ang,0))
                        + PSIJ_3D(ang,(ic-1),(k-1))
                        * HJ_1D(ang) * (1+HV_2D(ang,1))
                        + PSIK_3D(ang,(ic-1),(j-1))
                        * HK_1D(ang) * (1+HV_2D(ang,2));

                    if ( VDELT_CONST != 0 )
                    {
                        PC_1D(ang) += VDELT_CONST
                            * PTR_IN_6D(ang,(i-1),(j-1),(k-1),(i1-1),(i2-1))
                            * (1+HV_2D(ang,3));
                    }

                    PC_1D(ang) = PSI_1D(ang) + 0.5*PC_1D(ang);

                    DEN_1D(ang) = T_XS_3D((i-1),(j-1),(k-1))
                        + MU_1D(ang)  * HI * HV_2D(ang,0)
                        + HJ_1D(ang)  * HV_2D(ang,1)
                        + HK_1D(ang)  * HV_2D(ang,2)
                        + VDELT_CONST * HV_2D(ang,3);

                    if ( DEN_1D(ang) > TOLR )
                    {
                        PC_1D(ang) /= DEN_1D(ang);
                    }
                    else
                    {
                        PC_1D(ang) = 0;
                    }
                }
            } // end fixup_loop

            // Fixup done, compute edges
            for (ang = 0; ang < nang; ang++)
            {
                PSI_1D(ang) = PC_1D(ang);

                PSII_3D(ang,(j-1),(k-1)) = FXHV_2D(ang,0) * HV_2D(ang,0);

                PSIJ_3D(ang,(ic-1),(k-1)) = FXHV_2D(ang,1) * HV_2D(ang,1);

                if (NDIMEN == 3)
                {
                    PSIK_3D(ang,(ic-1),(j-1)) = FXHV_2D(ang,2) * HV_2D(ang,2);
                }

                if (VDELT_CONST != 0)
                {
                    PTR_OUT_6D(ang,(i-1),(j-1),(k-1),(i1-1),(i2-1)) = FXHV_2D(ang,3) * HV_2D(ang,3);
                }
            }
        }

        // Clear the flux arrays
        if ( oct == 1 )
        {
            FLUX_4D((i-1),(j-1),(k-1),(g-1)) = 0;

            for ( indx1 = 0; indx1 < (CMOM-1); indx1++ )
            {
                FLUXM_5D(indx1,(i-1),(j-1),(k-1),(g-1)) = 0;
            }
        }

        // Compute the flux moments
        sum_wpsi = 0;

        for (ang = 0; ang < nang; ang++)
        {
            sum_wpsi += W_1D(ang) * PSI_1D(ang);
        }

        FLUX_4D((i-1),(j-1),(k-1),(g-1)) += sum_wpsi;

        for (l = 1; l <= (CMOM-1); l++)
        {
            sum_ecwpsi = 0;

            for ( ang = 0; ang < nang; ang++ )
            {
                sum_ecwpsi += EC_2D(ang,(l))*W_1D(ang)*PSI_1D(ang);
            }

            FLUXM_5D((l-1),(i-1),(j-1),(k-1),(g-1)) += sum_ecwpsi;
        }

        // Calculate min and max scalar fluxes (not used elsewhere currently)
        if (oct == NOCT)
        {
            FMIN = MIN( FMIN, FLUX_3D((i-1),(j-1),(k-1)) );
            FMAX = MAX( FMAX, FLUX_3D((i-1),(j-1),(k-1)) );
        }

        // Save edge fluxes (dummy if checks for unused non-vacuum BCs)
        if (j == j_high)
        {
            if ((j_dir==2 && lasty) ||
                ((j_dir == 1 && firsty) && ibb == 1))
            {
                // CONTINUE
            }
            else
            {
                for (ang = 0; ang < nang; ang++)
                {
                    JB_OUT_3D(ang,(ic-1),(k-1)) = PSIJ_3D(ang,(ic-1),(k-1));
                }
            }
        }
        
        if (k == k_high)
        {
            if ((k_dir == 2 && lastz) ||
                ((k_dir==1 && firstz) && ibf == 1))
            {
                // CONTINUE
            }
            else
            {
                for ( ang = 0; ang < nang; ang++ )
                {
                    KB_OUT_3D(ang,(ic-1),(j-1)) = PSIK_3D(ang,(ic-1),(j-1));
                }
            }
        }

        // Compute leakages (not used elsewhere currently)
        if (((i+i_dir-1) == 1) || ((i+i_dir-1) == (NX+1)))
        {
            sum_wmupsii = 0;

            for (ang = 0; ang < nang; ang++)
            {
                sum_wmupsii += WMU_1D(ang) * PSII_3D(ang,(j-1),(k-1));
            }

            FLKX_3D((i+i_dir-1-1),(j-1),(k-1)) += i_step*sum_wmupsii;
        }
        if ((j_dir == 1 && firsty) || (j_dir == 2 && lasty))
        {
            sum_wetapsij = 0;

            for (ang = 0; ang < nang; ang++)
            {
                sum_wetapsij
                    += WETA_1D(ang) * PSIJ_3D(ang,(ic-1),(k-1));
            }

            FLKY_3D((i-1),(j+j_dir-1-1),(k-1)) += j_step*sum_wetapsij;
        }

        if (((k_dir == 1 && firstz) || (k_dir == 2 && lastz)) && NDIMEN == 3)
        {
            sum_wxipsik = 0;

            for (ang = 0; ang < nang; ang++)
            {
                sum_wxipsik += WXI_1D(ang) * PSIK_3D(ang,(ic-1),(j-1));
            }

            FLKZ_3D((i-1),(j-1),(k+k_dir-1-1))
                += k_step*sum_wxipsik;
        }
    }
}

void dim3_sweep_cuda ( input_data *input_vars, 
    bool firsty, bool lasty, bool firstz, bool lastz,
    geom_data *geom_vars, sn_data *sn_vars,
    data_data *data_vars, control_data *control_vars,
    solvar_data *solvar_vars, dim_sweep_data *dim_sweep_vars,
    int ich, int i_dir, int d1, int d2, int d3, int d4, int j_dir,
    int k_dir, int j_low, int k_low, int j_high, int k_high, int j_step, int k_step,
    int i1, int i2, int oct, int g, int *ierr )
{
    // Local variables
    int i;
    int ang, y_ind, ic_ind, z_ind = 4;

    double psi[NANG], pc[NANG], den[NANG];
    double hv[NANG*4], fxhv[NANG*4];

    double *c_psi[NANG], *c_pc[NANG], *c_den[NANG];
    double *c_hv[NANG*4], *c_fxhv[NANG*4];
    
    // Create GPU-copies of data
    input_data *c_input_vars;
    geom_data *c_geom_vars;
    sn_data *c_sn_vars;
    data_data *c_data_vars;
    control_data *c_control_vars;
    solvar_data *c_solvar_vars;
    dim_sweep_data *c_dim_sweep_vars;

    hipMalloc(&c_input_vars, sizeof(input_data));
    hipMalloc(&c_geom_vars, sizeof(geom_data));
    hipMalloc(&c_sn_vars, sizeof(sn_data));
    hipMalloc(&c_data_vars, sizeof(data_data));
    hipMalloc(&c_control_vars, sizeof(control_data));
    hipMalloc(&c_solvar_vars, sizeof(solvar_data));
    hipMalloc(&c_dim_sweep_vars, sizeof(dim_sweep_data));

    hipMalloc(c_psi, NANG * sizeof(double));
    hipMalloc(c_pc, NANG * sizeof(double));
    hipMalloc(c_den, NANG * sizeof(double));
    hipMalloc(c_hv, NANG * 4 * sizeof(double));
    hipMalloc(c_fxhv, NANG * 4 * sizeof(double));


    // Zero out the outgoing boundary arrays and fixup array
    for ( z_ind = 0; z_ind < NZ; z_ind++ )
    {
        for ( ic_ind = 0; ic_ind < ICHUNK; ic_ind++ )
        {
            for ( ang = 0; ang < NANG; ang++ )
            {
                JB_OUT_3D(ang,ic_ind,z_ind) = 0;
            }
        }
    }

    for ( y_ind = 0; y_ind < NY; y_ind++ )
    {
        for ( ic_ind = 0; ic_ind < ICHUNK; ic_ind++ )
        {
            for ( ang = 0; ang < NANG; ang++ )
            {
                KB_OUT_3D(ang,ic_ind,y_ind) = 0;
            }
        }
    }

    for ( i = 0; i < 4; i++)
    {
        for ( ang = 0; ang < NANG; ang++ )
        {
            FXHV_2D(ang, i) = 0;
        }
    }

    
    hipMemcpy(c_psi, psi, NANG * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(c_pc, pc, NANG * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(c_den, den, NANG * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(c_hv, hv, NANG * 4 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(c_fxhv, fxhv, NANG * 4 * sizeof(double), hipMemcpyHostToDevice);

    hipMemcpy(c_input_vars, input_vars, sizeof(input_data), hipMemcpyHostToDevice);
    hipMemcpy(c_geom_vars, geom_vars, sizeof(geom_data), hipMemcpyHostToDevice);
    hipMemcpy(c_sn_vars, sn_vars, sizeof(sn_data), hipMemcpyHostToDevice);
    hipMemcpy(c_data_vars, data_vars, sizeof(data_data), hipMemcpyHostToDevice);
    hipMemcpy(c_control_vars, control_vars, sizeof(control_data), hipMemcpyHostToDevice);
    hipMemcpy(c_solvar_vars, solvar_vars, sizeof(solvar_data), hipMemcpyHostToDevice);
    hipMemcpy(c_dim_sweep_vars, dim_sweep_vars, sizeof(dim_sweep_data), hipMemcpyHostToDevice);

    // Loop over cells along the diagonals. When only 1 diagonal, it's
    // normal sweep order. Otherwise, nested threading performs mini-KBA.
    // diagonal loop
    int d;
    for (d = 1; d <= NDIAG; d++)
    {
        printf("lenc: %d\n", DIAG_1D(d-1).lenc);
        diagonal_loop<<<1, (DIAG_1D(d-1).lenc)>>>(
            c_input_vars,
            firsty, lasty, firstz, lastz,
            c_geom_vars, c_sn_vars,
            c_data_vars, c_control_vars,
            c_solvar_vars, c_dim_sweep_vars,

            ich, i_dir, d1, d2, d3, d4, j_dir,
            k_dir, j_low, k_low, j_high, k_high, j_step, k_step,
            i1, i2, oct, g, ierr,

            input_vars->nang,
            *c_psi, *c_pc, *c_den, *c_hv, *c_fxhv,
            d
        );
    }

    // Copy from device back to hsot
    hipMemcpy(c_input_vars, input_vars, sizeof(input_data), hipMemcpyDeviceToHost);
    hipMemcpy(c_geom_vars, geom_vars, sizeof(geom_data), hipMemcpyDeviceToHost);
    hipMemcpy(c_sn_vars, sn_vars, sizeof(sn_data), hipMemcpyDeviceToHost);
    hipMemcpy(c_data_vars, data_vars, sizeof(data_data), hipMemcpyDeviceToHost);
    hipMemcpy(c_control_vars, control_vars, sizeof(control_data), hipMemcpyDeviceToHost);
    hipMemcpy(c_solvar_vars, solvar_vars, sizeof(solvar_data), hipMemcpyDeviceToHost);
    hipMemcpy(c_dim_sweep_vars, dim_sweep_vars, sizeof(dim_sweep_data), hipMemcpyDeviceToHost);

    // Clean up
    hipFree(c_input_vars);
    hipFree(c_geom_vars);
    hipFree(c_sn_vars);
    hipFree(c_data_vars);
    hipFree(c_control_vars);
    hipFree(c_solvar_vars);
    hipFree(c_dim_sweep_vars);

    hipFree(c_psi);
    hipFree(c_pc);
    hipFree(c_den);
    hipFree(c_hv);
    hipFree(c_fxhv);
}