#include <hip/hip_runtime.h>

#define N 10000000

__global__ void vector_add(float *out, float *a, float *b, int n)
{
	for (int i = 0; i < n; i++)
	{
		out[i] = a[i] + b[i];
	}
}

int main()
{
	float *a, *b, *out;
	float *d_a, *d_b;

	// Allocate memory
	a = (float*)malloc(sizeof(float) * N);
	b = (float*)malloc(sizeof(float) * N);
	out = (float*)malloc(sizeof(float) * N);

	hipMalloc((void**)&d_a, sizeof(float) * N);
	hipMalloc((void**)&d_b, sizeof(float) * N);

	hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

	// Init array
	for (int i = 0; i < N; i++)
	{
		a[i] = 1.0f;
		b[i] = 2.0f;
	}

	vector_add<<<1,1>>>(out, d_a, d_b, N);

	hipFree(d_a); free(a);
	hipFree(d_b); free(b);
}
